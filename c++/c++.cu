#include "hip/hip_runtime.h"
﻿#include <thrust/device_ptr.h>
#include <thrust/count.h>
#include <thrust/execution_policy.h>

#include <iostream>
#include <hip/hip_runtime_api.h>

/////////////////////////////////////////////////////////////////
// Some utility code to define grid_stride_range
// Normally this would be in a header but it's here
// for didactic purposes. Uses
#include "range.hpp"
using namespace util::lang;

// type alias to simplify typing...
template <typename T>
using step_range = typename range_proxy<T>::step_range_proxy;
//获得一个可迭代的范围对象
template <typename T>
__device__ step_range<T> grid_stride_range(T begin, T end) {
    begin += blockDim.x * blockIdx.x + threadIdx.x;
    return range(begin, end).step(gridDim.x * blockDim.x);
}
/////////////////////////////////////////////////////////////////
//计算长度为n的数组data中满足给定谓词p的元素数量. Predicate p 谓词/断言（即判断条件）
template <typename T, typename Predicate>
__device__ void count_if(int* count, T* data, int n, Predicate p) {
    for (auto i : grid_stride_range(0, n)) {
        if (p(data[i])) atomicAdd(count, 1);
    }
}

// Use count_if with a lambda function that searches for x, y, z or w
// Note the use of range-based for loop and initializer_list inside the functor
// We use auto so we don't have to know the type of the functor or array
__global__ void xyzw_frequency(int* count, char* text, int n) {
    const char letters[]{ 'x', 'y', 'z', 'w' };

    count_if(count, text, n, [&](char c) {  //函数签名：count_if(first, last, pred)
        for (const auto x : letters)  //lamba函数 用于遍历容器中的元素
            if (c == x) return true;
        return false;
        });
}

__global__ void xyzw_frequency_thrust_device(int* count, char* text, int n) {
    const char letters[]{ 'x', 'y', 'z', 'w' };
    //Thrust 库中的 count_if 算法 将字符数组 text 从 CPU 内存复制到 GPU 设备内存上；
    *count = thrust::count_if(thrust::device, text, text + n, [=](char c) {
        for (const auto x : letters)
            if (c == x) return true;
        return false;
        });
}

// a bug in Thrust 1.8 causes warnings when this is uncommented
// so commented out by default -- fixed in Thrust master branch
#if 0 
void xyzw_frequency_thrust_host(int* count, char* text, int n)
{
    const char letters[]{ 'x','y','z','w' };
    *count = thrust::count_if(thrust::host, text, text + n, [&](char c) {
        for (const auto x : letters)
            if (c == x) return true;
        return false;
        });
}
#endif

int main(int argc, char** argv) {
    const char* filename = sdkFindFilePath("warandpeace.txt", argv[0]);

    int numBytes = 16 * 1048576;
    char* h_text = (char*)malloc(numBytes);

    // find first CUDA device
    int devID = findCudaDevice(argc, (const char**)argv);

    char* d_text;                                                   //cuda text
    checkCudaErrors(hipMalloc((void**)&d_text, numBytes));

    FILE* fp = fopen(filename, "r");
    if (fp == NULL) {
        printf("Cannot find the input text file\n. Exiting..\n");
        return EXIT_FAILURE;
    }
    int len = (int)fread(h_text, sizeof(char), numBytes, fp);
    fclose(fp);
    std::cout << "Read " << len << " byte corpus from " << filename << std::endl;

    checkCudaErrors(hipMemcpy(d_text, h_text, len, hipMemcpyHostToDevice));  //向左拷贝

    int count = 0;
    int* d_count;
    checkCudaErrors(hipMalloc(&d_count, sizeof(int)));  //申请内存
    checkCudaErrors(hipMemset(d_count, 0, sizeof(int)));  // 计数器置零

    // Try uncommenting one kernel call at a time
    xyzw_frequency << <8, 256 >> > (d_count, d_text, len);
    xyzw_frequency_thrust_device << <1, 1 >> > (d_count, d_text, len);
    checkCudaErrors(
        hipMemcpy(&count, d_count, sizeof(int), hipMemcpyDeviceToHost));

    // xyzw_frequency_thrust_host(&count, h_text, len);

    std::cout << "counted " << count
        << " instances of 'x', 'y', 'z', or 'w' in \"" << filename << "\""
        << std::endl;

    checkCudaErrors(hipFree(d_count));
    checkCudaErrors(hipFree(d_text));

    return EXIT_SUCCESS;
}
